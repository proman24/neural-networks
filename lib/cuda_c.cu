#include "hip/hip_runtime.h"
/**
 *  CUDA PARALLEL PROGRAMMING: matrix_ops.cu
 *  Purpose: Matrix Operations using CUDA C/C++
 *  @author Prabhsimran Singh
 *  @version 1.0 17/09/18
 *
 *  Build using: nvcc -Xcompiler -fPIC -shared -o lib/cuda_mat_ops.so matrix_ops.cu
 */

#include <iostream>
#include <math.h>
#include "utils/devices.cu"
#include "utils/utils.cpp"

#define BLOCK_SIZE 256

/**
* Calculates dot-product of two matrices (using parallel threads on CUDA capable device)
*
* @param a the float pointer to first input array
* @param b the float pointer to second input array
* @param c the float pointer to output array
* @param m the no. rows in a(m x n) and c(m x k)
* @param n the no. cols in a(m x n) and rows in b(n x k)
* @param k the no. cols in b(n x k) and c(m x k)
* @return void
*/
__global__ void matmul(float *a, float *b, float *c, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0;

    if (row < m && col < k) {
        for (int i = 0; i < n; i++)
            sum += a[row * n + i] * b[i * k + col];
        c[row * k + col] = sum;
    }
}

/**
* Calculates element-wise sum of two matrices (using parallel threads on CUDA capable device)
*
* @param a the float pointer to first input array
* @param b the float pointer to second input array
* @param c the float pointer to output array
* @param m the no. of rows in the arrays
* @param n the no. of cols in the arrays
* @return void
*/
__global__ void matsum(float *a, float *b, float *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n)
        c[row * n + col] = a[row * n + col] + b[row * n + col];
}

/**
* Calculates element-wise product of two matrices (using parallel threads on CUDA capable device)
*
* @param a the float pointer to first input array
* @param b the float pointer to second input array
* @param c the float pointer to output array
* @param m the no. of rows in the arrays
* @param n the no. of cols in the arrays
* @return void
*/
__global__ void matprod(float *a, float *b, float *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n)
        c[row * n + col] = a[row * n + col] * b[row * n + col];
}

/**
* Calculates element-wise sum of a matrix with a value (using parallel threads on CUDA capable device)
*
* @param a the float pointer to first input array
* @param b the float value to add the array with
* @param c the float pointer to output array
* @param m the no. of rows in the arrays
* @param n the no. of cols in the arrays
* @return void
*/
__global__ void elemwise_sum(float *a, float b, float *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n)
        c[row * n + col] = a[row * n + col] + b;
}

/**
* Calculates element-wise product of a matrix with a value (using parallel threads on CUDA capable device)
*
* @param a the float pointer to first input array
* @param b the float value to multiply the array with
* @param c the float pointer to output array
* @param m the no. of rows in the arrays
* @param n the no. of cols in the arrays
* @return void
*/
__global__ void elemwise_prod(float *a, float b, float *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n)
        c[row * n + col] = a[row * n + col] * b;
}

/**
* Calculates element-wise maximum of a matrix with a value (using parallel threads on CUDA capable device)
*
* @param a the float pointer to first input array
* @param b the float value to check maximum against
* @param c the float pointer to output array
* @param m the no. of rows in the arrays
* @param n the no. of cols in the arrays
* @return void
*/
__global__ void elemwise_max(float *a, float b, float *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n)
        c[row * n + col] = (a[row * n + col] > b) ? a[row * n + col] : b;
}

extern "C" {

    void cuda_matmul(float *a, float *b, float *c, int m, int n, int k) {
        float *d_a, *d_b, *d_c;

        hipMallocManaged(&d_a, (m * n) * sizeof(float));
        hipMallocManaged(&d_b, (n * k) * sizeof(float));
        hipMallocManaged(&d_c, (m * k) * sizeof(float));

        hipMemcpy(d_a, a, (m * n) * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, (n * k) * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = m / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        matmul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
        hipDeviceSynchronize();
    
        hipMemcpy(c, d_c, (m * k) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }

    void cuda_matsum(float *a, float *b, float *c, int m, int n) {
        float *d_a, *d_b, *d_c;

        hipMallocManaged(&d_a, (m * n) * sizeof(float));
        hipMallocManaged(&d_b, (m * n) * sizeof(float));
        hipMallocManaged(&d_c, (m * n) * sizeof(float));

        hipMemcpy(d_a, a, (m * n) * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, (m * n) * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = m / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        matsum<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, (m * n) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }

    void cuda_matprod(float *a, float *b, float *c, int m, int n) {
        float *d_a, *d_b, *d_c;

        hipMallocManaged(&d_a, (m * n) * sizeof(float));
        hipMallocManaged(&d_b, (m * n) * sizeof(float));
        hipMallocManaged(&d_c, (m * n) * sizeof(float));

        hipMemcpy(d_a, a, (m * n) * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, (m * n) * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = m / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        matprod<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, (m * n) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }

    void cuda_elemwise_sum(float *a, float b, float *c, int m, int n) {
        float *d_a, *d_c;

        hipMallocManaged(&d_a, (m * n) * sizeof(float));
        hipMallocManaged(&d_c, (m * n) * sizeof(float));

        hipMemcpy(d_a, a, (m * n) * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = m / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        elemwise_sum<<<dimGrid, dimBlock>>>(d_a, b, d_c, m, n);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, (m * n) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_c);
    }

    void cuda_elemwise_prod(float *a, float b, float *c, int m, int n) {
        float *d_a, *d_c;

        hipMallocManaged(&d_a, (m * n) * sizeof(float));
        hipMallocManaged(&d_c, (m * n) * sizeof(float));

        hipMemcpy(d_a, a, (m * n) * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = m / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        elemwise_prod<<<dimGrid, dimBlock>>>(d_a, b, d_c, m, n);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, (m * n) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_c);
    }
 
    void cuda_elemwise_max(float *a, float b, float *c, int m, int n) {
        float *d_a, *d_c;

        hipMallocManaged(&d_a, (m * n) * sizeof(float));
        hipMallocManaged(&d_c, (m * n) * sizeof(float));

        hipMemcpy(d_a, a, (m * n) * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = m / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        elemwise_max<<<dimGrid, dimBlock>>>(d_a, b, d_c, m, n);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, (m * n) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_c);
    }
 }